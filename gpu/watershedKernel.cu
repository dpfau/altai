#include "hip/hip_runtime.h"
/*
 * Kernel to compute watershed seeded from one pixel
 *
 * David Pfau, 2014
 */

#define MEM_BLOCK 512
#include <stdio.h>

__device__ int * myrealloc(int * old, int oldsize, int newsize)
{
    int * newT = (int *) malloc (newsize*sizeof(int));

    for(int i=0; i<oldsize; i++)
    {
        newT[i] = old[i];
    }

    free(old);
    return newT;
}

__device__ int checkIndex(const int idx, const int * offset, const int * dims, const int ndims) {
    switch (ndims) {
        case 2:
            return (idx % dims[0]) + offset[0] >= 0 && (idx % dims[0]) + offset[0] < dims[0]
                && (idx / dims[0]) + offset[1] >= 0 && (idx / dims[0]) + offset[1] < dims[1];
        case 3:
            return (idx % dims[0])             + offset[0] >= 0 && (idx % dims[0])             + offset[0] < dims[0]
                && ((idx / dims[0]) % dims[1]) + offset[1] >= 0 && ((idx / dims[0]) % dims[1]) + offset[1] < dims[1] 
                && (idx / (dims[0] * dims[1])) + offset[2] >= 0 && (idx / (dims[0] * dims[1])) + offset[2] < dims[2];
    }
    return 0;
}

__device__ int offsetIndex(const int idx, const int * offset, const int * dims, const int ndims) {
    switch (ndims) {
        case 2:
            return (idx % dims[0]) + offset[0] + dims[0] * ((idx / dims[0]) + offset[1]);
        case 3:
            return                  (idx % dims[0])              + offset[0]  + 
                dims[0] *           (((idx / dims[0]) % dims[1]) + offset[1]) + 
                dims[0] * dims[1] * ((idx / (dims[0] * dims[1])) + offset[2]);
    }
    return 0;
}

__device__ int checkOneNeighbor(const float * A, int idx, int old_idx, const int * offset, const int * dims, const int ndims) {
    if ( checkIndex(idx, offset, dims, ndims) ) {
        int new_idx = offsetIndex(idx, offset, dims, ndims);
        if (A[new_idx] > A[old_idx]) {
             return new_idx;
        } else {
          return old_idx;
        }
    }
    return old_idx; 
}

__device__ int checkOneIndex(const float * A, int * idx, int q, int * offset, const int * dims, const int ndims, int npix, int dpix) {
    if ( checkIndex(idx[q], offset, dims, ndims) ) { // check that index is within bounds
        int new_idx = offsetIndex(idx[q], offset, dims, ndims);
        for (int qq=0; qq<npix; qq++) { 
        // Check that index is not in the list. 
        // Linear search is stupid and slow, but I'm not sure how to put a hash table inside CUDA device code
            if (idx[qq]==new_idx) {
                new_idx = -1;
                break;
            }
        }
        if (new_idx != -1) {
            if (A[new_idx] > 0.0f) {
                int neighbor_idx = new_idx;
                for (int iii=-1; iii<=1; iii++) {
                    for (int jjj=-1; jjj<=1; jjj++) {
                       offset[0] = iii; offset[1] = jjj;
                       switch (ndims) {
                           case 2:
                               neighbor_idx = checkOneNeighbor(A, new_idx, neighbor_idx, offset, dims, ndims);
                           case 3:
                           for (int kkk=-1; kkk<=1; kkk++) {
                               offset[2] = kkk;
                               neighbor_idx = checkOneNeighbor(A, new_idx, neighbor_idx, offset, dims, ndims);
                           }
                       }
                    }
                }
                for (int qq=0; qq < npix; qq++) {
                    if (idx[qq]==neighbor_idx) {
                        if ( (npix + dpix) % MEM_BLOCK == 0 ) {
                            idx = myrealloc(idx, npix + dpix, npix + dpix + MEM_BLOCK);
                        }
                        idx[npix + dpix++] = new_idx;
                        break;
                    }
                }
            }
        }
    }
    return dpix;
}

__global__ void watershedKernel(const float * A, int * B, const int * seedIdx, const int N, const int ndims, const int * dims)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < N) {
        int * idx = (int *)malloc(sizeof(int) * MEM_BLOCK); // This array will grow dynamically as we add new indices to the list
        int npix = 1; // number of pixels in the watershed
        int dpix = 1; // change in number of pixels in this loop
        idx[0] = seedIdx[i];
        while (dpix > 0) {
            printf("\n%d",npix);
            dpix = 0;
            for (int q=0; q<npix; q++) {
               for (int ii=-1; ii<=1; ii++) {
                  for (int jj=-1; jj<=1; jj++) {
                      switch(ndims) {
                          case 2:
                              int offset[] = {ii, jj};
                              dpix = checkOneIndex(A, idx, q, offset, dims, ndims, npix, dpix);
                          case 3:
                              for (int kk=-1; kk<=1; kk++) {
                                  int offset[] = {ii, jj, kk};
                                  dpix = checkOneIndex(A, idx, q, offset, dims, ndims, npix, dpix);
                              }
                      }
                  }
               }
            }
            npix += dpix;
        }

        for (int k=0; k < npix; k++) {
            B[idx[k]] = i;
        }

        free(idx);
    }
}
