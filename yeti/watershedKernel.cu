#include "hip/hip_runtime.h"
/*
 * Kernel to compute watershed seeded from one pixel
 *
 * David Pfau, 2014
 */

#define MEM_BLOCK 512

__global__ void watershedKernel(const float * A, int * B, const int * seedIdx, const int N, const int ndims, const int * dims)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < N) {
        int * idx = (int *)malloc(sizeof(int) * MEM_BLOCK); // This array will grow dynamically as we add new indices to the list
        int npix = 1; // number of pixels in the watershed
        int dpix = 1; // change in number of pixels in this loop
        idx[0] = seedIdx[i];
        while (dpix > 0) {
            dpix = 0;
            for (int k=0; k<npix; k++) {
               for (int ii=-1; ii<=1; ii++) {
                  for (int jj=-1; jj<=1; jj++) {
                      switch(ndims) {
                          case 2:
                              if ( (idx[k] % dims[0]) + ii >= 0 && (idx[k] % dims[0]) + ii < dims[0]
                                && (idx[k] / dims[0]) + jj >= 0 && (idx[k] / dims[0]) + jj < dims[1] ) { // check that index is within bounds
                                  int new_idx = (idx[k] % dims[0]) + ii + dims[0] * ((idx[k] / dims[0]) + jj);
                                  for (int kk=0; kk<npix; kk++) { 
                                  // Check that index is not in the list. 
                                  // Linear search is stupid and slow, but I'm not sure how to put a hash table inside CUDA device code
                                      if (idx[kk]==new_idx) {
                                          new_idx = -1;
                                          break;
                                      }
                                  }
                                  if (new_idx != -1) {
                                      if (A[new_idx] > 0.0f) {
                                          int neighbor_idx = new_idx;
                                          for (int iii=-1; iii<=1; iii++) {
                                              for (int jjj=-1; jjj<=1; jjj++) {
                                                 if ( (new_idx % dims[0]) + iii >= 0 && (new_idx % dims[0]) + iii < dims[0]
                                                   && (new_idx / dims[0]) + jjj >= 0 && (new_idx / dims[0]) + jjj < dims[1] ) {
                                                    int this_idx = (new_idx % dims[0]) + iii + dims[0] * ((new_idx / dims[0]) + jjj);
                                                    if (A[this_idx] > A[neighbor_idx]) {
                                                        neighbor_idx = this_idx;
                                                    }
                                                 } 
                                              }
                                          }
                                          for (int kk=0; kk < npix; kk++) {
                                              if (idx[kk]==neighbor_idx) {
                                                  if ( (npix + dpix) % MEM_BLOCK == 0 ) {
                                                      int * bigger_idx = (int *)realloc(idx, npix + dpix + MEM_BLOCK);
                                                      if (bigger_idx != NULL) {
                                                          idx = bigger_idx;
                                                      } else {
                                                          free(idx);
                                                          puts("Error reallocating memory");
                                                          exit(1);
                                                      }
                                                  }
                                                  idx[npix + dpix++] = new_idx;
                                                  break;
                                              }
                                          }
                                      }
                                  }
                              }
                          case 3:
                      }
                  }
               }
            }
            npix += dpix;
        }

        for (int k=0; k < npix; k++) {
            B[idx[k]] = i;
        }

        free(idx);
    }
}
