
#include <hip/hip_runtime.h>
/*
 * Kernel to compute watershed seeded from one pixel
 *
 * David Pfau, 2014
 */

__global__ void watershedKernel(float * A, float * B, const int * seedIdx, const int N, const int ndims, const int * dims)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < N) {
        B[i] = 2.0f * A[i];
    }
}