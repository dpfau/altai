#include "hip/hip_runtime.h"
/*
 * Kernel to compute watershed seeded from one pixel
 *
 * David Pfau, 2014
 */

void __global__ watershedKernel(const float * A, const float * B, const int * seedIdx, const int N, const int ndims, const int * dims)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < N) {
        B[i] = 2.0 * A[i];
    }
}